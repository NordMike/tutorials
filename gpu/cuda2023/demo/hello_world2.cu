// Copyright 2023 Pierre Talbot


#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDIE(result) { \
  hipError_t e = (result); \
  if (e != hipSuccess) { \
    printf("%s:%d CUDA runtime error %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
  }}

__host__ __device__ void print(const char* msg) {
  printf("%s\n", msg);
}

__global__ void hello_world() {
  print("world");
}

int main(int argc, char** argv) {
  print("hello");
  hello_world<<<1, 1>>>();
  CUDIE(hipDeviceSynchronize())
  return 0;
}
